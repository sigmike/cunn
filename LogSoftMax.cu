#include "hip/hip_runtime.h"
#define MINUS_LOG_THRESHOLD -18.42
#define LOGSOFTMAX_THREADS 128

struct addvalue_functor
{
  const float value;

  addvalue_functor(float value_) : value(value_) {}

    __host__ __device__ float operator()(const float& x) const
  {
    return (x+value);
  }
};

__global__ void cunn_LogSoftMax_updateOutput_kernel(float *output, float *input, int nframe, int dim)
{
  __shared__ float buffer[LOGSOFTMAX_THREADS+1];
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *output_k = output + k*dim;

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  // max?
  buffer[threadIdx.x] = -FLT_MAX;
  for (int i=i_start; i<i_end; i+=i_step)
  {
    float z = input_k[i];
    if(buffer[threadIdx.x] < z)
      buffer[threadIdx.x] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float max_k = -FLT_MAX;
    for (int i=0; i<blockDim.x; i++)
    {
      if(max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[LOGSOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // logadd?
  float max_k = buffer[LOGSOFTMAX_THREADS];
  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step)
    buffer[threadIdx.x] += __expf(input_k[i]-max_k);

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float logsum_k = 0;
    for (int i=0; i<blockDim.x; i++)
      logsum_k += buffer[i];
    buffer[LOGSOFTMAX_THREADS] = max_k + __logf(logsum_k);
  }

  __syncthreads();

  // logsoftmax
  float logsum_k = buffer[LOGSOFTMAX_THREADS];
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i] = input_k[i] - logsum_k;
}


__global__ void cunn_LogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
  __shared__ float buffer[LOGSOFTMAX_THREADS];
  int k = blockIdx.x;
  float *gradInput_k = gradInput + k*dim;
  float *output_k = output + k*dim;
  float *gradOutput_k = gradOutput + k*dim;

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  // sum?
  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step)
    buffer[threadIdx.x] += gradOutput_k[i];

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float sum_k = 0;
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[0] = sum_k;
  }

  __syncthreads();

  float sum_k = buffer[0];
  for (int i=i_start; i<i_end; i+=i_step)
    gradInput_k[i] = gradOutput_k[i] - __expf(output_k[i])*sum_k;
}

static int cunn_LogSoftMax_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  input = THCudaTensor_newContiguous(input);
  THCudaTensor_resizeAs(output, input);

  if(input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(LOGSOFTMAX_THREADS);
    cunn_LogSoftMax_updateOutput_kernel<<<blocks,threads>>>(THCudaTensor_data(output), THCudaTensor_data(input), 1, input->size[0]);
  }
  else if(input->nDimension == 2)
  {
    dim3 blocks(input->size[0]);
    dim3 threads(LOGSOFTMAX_THREADS);
    cunn_LogSoftMax_updateOutput_kernel<<<blocks,threads>>>(THCudaTensor_data(output), THCudaTensor_data(input), input->size[0], input->size[1]);
  }
  else
    THError("vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(input);
  return 1;
}

struct logsoftmaxupdateGradInput_functor
{
  float value;

  logsoftmaxupdateGradInput_functor(float value_) : value(value_) {}

  __host__ __device__ float operator()(const float& output, const float& gradOutput) const
  {
    return gradOutput - exp(output)*value;
  }
};

static int cunn_LogSoftMax_updateGradInput(lua_State *L)
{
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  output = THCudaTensor_newContiguous(output);
  gradOutput = THCudaTensor_newContiguous(gradOutput);

  THCudaTensor_resizeAs(gradInput, output);

  if(gradInput->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(LOGSOFTMAX_THREADS);

    cunn_LogSoftMax_updateGradInput_kernel<<<blocks,threads>>>(THCudaTensor_data(gradInput),
                                                        THCudaTensor_data(output),
                                                        THCudaTensor_data(gradOutput),
                                                        1, gradInput->size[0]);
  }
  else if(gradInput->nDimension == 2)
  {
    dim3 blocks(gradInput->size[0]);
    dim3 threads(LOGSOFTMAX_THREADS);

    cunn_LogSoftMax_updateGradInput_kernel<<<blocks,threads>>>(THCudaTensor_data(gradInput),
                                                        THCudaTensor_data(output),
                                                        THCudaTensor_data(gradOutput),
                                                        gradInput->size[0], gradInput->size[1]);
  }
  else
    THError("vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(gradOutput);
  THCudaTensor_free(output);
  return 1;
}

static const struct luaL_Reg cunn_LogSoftMax__ [] = {
  {"LogSoftMax_updateOutput", cunn_LogSoftMax_updateOutput},
  {"LogSoftMax_updateGradInput", cunn_LogSoftMax_updateGradInput},
  {NULL, NULL}
};

static void cunn_LogSoftMax_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_LogSoftMax__, "nn");
  lua_pop(L,1);
}
