#include "hip/hip_runtime.h"
struct sigmoidupdateOutput_functor
{
  __host__ __device__ float operator()(const float& input) const
  {
    return 1./(1.+ exp(-input));
  }
};

static int cunn_Sigmoid_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  long size = THCudaTensor_nElement(input);

  input = THCudaTensor_newContiguous(input);

  THCudaTensor_resizeAs(output, input);

  thrust::device_ptr<float> output_data(THCudaTensor_data(output));
  thrust::device_ptr<float> input_data(THCudaTensor_data(input));
  thrust::transform(input_data, input_data+size, output_data, sigmoidupdateOutput_functor());

  THCudaTensor_free(input);
  return 1;
}

struct sigmoidupdateGradInput_functor
{
  __host__ __device__ float operator()(const float& output, const float& gradOutput) const
  {
    return gradOutput * (1.-output) * output;
  }
};

static int cunn_Sigmoid_updateGradInput(lua_State *L)
{
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  long size = THCudaTensor_nElement(output);

  gradOutput = THCudaTensor_newContiguous(gradOutput);

  THCudaTensor_resizeAs(gradInput, output);

  thrust::device_ptr<float> output_data(THCudaTensor_data(output));
  thrust::device_ptr<float> gradOutput_data(THCudaTensor_data(gradOutput));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(gradInput));
  thrust::transform(output_data, output_data+size, gradOutput_data, gradInput_data, sigmoidupdateGradInput_functor());

  THCudaTensor_free(gradOutput);
  return 1;
}

static const struct luaL_Reg cunn_Sigmoid__ [] = {
  {"Sigmoid_updateOutput", cunn_Sigmoid_updateOutput},
  {"Sigmoid_updateGradInput", cunn_Sigmoid_updateGradInput},
  {NULL, NULL}
};

static void cunn_Sigmoid_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Sigmoid__, "nn");
  lua_pop(L,1);
}
